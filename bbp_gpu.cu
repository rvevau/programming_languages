#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NDEC 16
#define BLOCK_SIZE 256  // Размер блока для CUDA

// Функция для вычисления десятичных цифр числа Pi
__device__ double expm_cuda(double p, double ak)
{
    int i, j;
    double p1, pt, r;
#define ntp 25
    static double tp[ntp];
    static int tp1 = 0;

    if (tp1 == 0) {
        tp1 = 1;
        tp[0] = 1.;

        for (i = 1; i < ntp; i++) tp[i] = 2. * tp[i - 1];
    }

    if (ak == 1.) return 0.;

    for (i = 0; i < ntp; i++) if (tp[i] > p) break;

    pt = tp[i - 1];
    p1 = p;
    r = 1.;

    for (j = 1; j <= i; j++) {
        if (p1 >= pt) {
            r = 16. * r;
            r = r - (int)(r / ak) * ak;
            p1 = p1 - pt;
        }
        pt = 0.5 * pt;
        if (pt >= 1.) {
            r = r * r;
            r = r - (int)(r / ak) * ak;
        }
    }

    return r;
}

// Функция для вычисления членов BBP-суммы для конкретного m
__global__ void compute_series(int m, int id, double* results)
{
    int k = blockIdx.x * blockDim.x + threadIdx.x;
    double ak, p, s, t;
    if (k < id) {
        ak = 8 * k + m;
        p = id - k;
        t = expm_cuda(p, ak);
        s = t / ak;
        results[k] = s;
    }
}

// Функция для вычисления Pi на основе суммы BBP
double series(int m, int id, double* d_results)
{
    double* h_results = (double*)malloc(id * sizeof(double));
    double s = 0.0;
    int num_blocks = (id + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Копируем данные на GPU
    hipMemcpy(d_results, h_results, id * sizeof(double), hipMemcpyHostToDevice);

    // Запускаем CUDA kernel для вычисления членов серии
    compute_series << <num_blocks, BLOCK_SIZE >> > (m, id, d_results);

    // Копируем результаты с GPU на CPU
    hipMemcpy(h_results, d_results, id * sizeof(double), hipMemcpyDeviceToHost);

    // Складываем результаты
    for (int k = 0; k < id; k++) {
        s += h_results[k];
        s -= (int)s;  // Оставляем только дробную часть
    }

    free(h_results);
    return s;
}

// Функция для преобразования числа Pi в десятичные цифры
void idec(double x, int ndec, char cdec[])
{
    int i;
    double y;

    y = fabs(x);

    for (i = 0; i < ndec; i++) {
        y = 10. * (y - floor(y));
        cdec[i] = '0' + (int)y;
    }
    cdec[ndec - 1] = '\0';
}

int main()
{
    double pid, s1, s2, s3, s4;
    double* d_results;
    int id = 100000000;
    char cdec[NDEC];

    // Выделение памяти для хранения результатов на GPU
    hipMalloc((void**)&d_results, id * sizeof(double));

    // Замер времени
    clock_t start_time, end_time;
    double time_taken;
    start_time = clock();

    // Вычисляем части формулы BBP с использованием CUDA
    s1 = series(1, id, d_results);
    s2 = series(4, id, d_results);
    s3 = series(5, id, d_results);
    s4 = series(6, id, d_results);
    pid = 4. * s1 - 2. * s2 - s3 - s4;
    pid = pid - (int)pid + 1.;

    // Преобразуем в десятичные цифры
    idec(pid, NDEC, cdec);

    // Выводим результат
    printf("Position: %i Decimal digits: %s\n", id, cdec);

    // Конец замера времени
    end_time = clock();
    time_taken = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;
    printf("Time taken: %f seconds\n", time_taken);

    // Освобождение памяти на GPU
    hipFree(d_results);

    return 0;
}
